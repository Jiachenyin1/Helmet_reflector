#include "hip/hip_runtime.h"
//#include "mish.h"
//#include <cmath>
//#include <stdio.h>
//#include <cassert>
//#include <iostream>

//namespace nvinfer1
//{
//    MishPlugin::MishPlugin()
//    {

//    }

//    MishPlugin::~MishPlugin(){}
//    MishPlugin::MishPlugin(const void* seriaData , size_t seriaLen)
//    {
//        assert(seriaLen == sizeof(input_size_));
//        input_size_ = *reinterpret_cast<const int*>(seriaData);
//    }

//    void MishPlugin::serialize(void *buffer) const
//    {
//        *reinterpret_cast<int* >(buffer) = input_size_;
//    }

//    size_t MishPlugin::getSerializationSize() const
//    {
//        return sizeof(input_size_);
//    }

//    int MishPlugin::initialize()
//    {
//        return 0;
//    }

//    Dims MishPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
//    {
//        assert(nbInputDims == 1);
//        assert(index == 0 );

//        input_size_ = inputs[0].d[0] * inputs[0].d[1] * inputs[0].d[2] ;
//        return Dims3(inputs[0].d[0] , inputs[0].d[1] , inputs[0].d[2]);
//    }

//    void MishPlugin::setPluginNamespace(const char* pluginNamespce)
//    {
//        mPluginNamespace = pluginNamespce;
//    }

//    const char* MishPlugin::getPluginNamespace() const
//    {
//        return mPluginNamespace;
//    }

//    DataType MishPlugin::getOutputDataType(int index, const DataType *inputTypes, int nbInputs) const
//    {
//        return DataType::kFLOAT;
//    }

//    bool MishPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool *inputIsBroadcasted, int nbInputs) const
//    {
//        return false;
//    }

//    bool MishPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
//    {
//        return false ;
//    }

//    void MishPlugin::configurePlugin(const PluginTensorDesc *in, int nbInput, const PluginTensorDesc *out, int nbOutput)
//    {

//    }
//    //一个都不能少。。。
//    void MishPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
//    {
//    }
//    void MishPlugin::detachFromContext() {}

//    const char* MishPlugin::getPluginType() const
//    {
//        return "Mish_TRT";
//    }

//    const char* MishPlugin::getPluginVersion() const
//    {
//        return "1";
//    }


//    void MishPlugin::destroy()
//    {
//        delete this ;
//    }

//    IPluginV2IOExt* MishPlugin::clone() const
//    {
//        MishPlugin* p = new MishPlugin();
//        p->input_size_ = input_size_ ;
//        p->setPluginNamespace(mPluginNamespace);
//        return p;
//    }
//    __device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}

//    __device__ float softplus_kernel(float x, float threshold = 20) {
//        if (x > threshold) return x;                // too large
//        else if (x < -threshold) return expf(x);    // too small
//        return logf(expf(x) + 1);
//    }

//    __global__ void mish_kernel(const float *input, float *output, int num_elem) {
//        //即tid=blockIdx.x（当前块的ID）*blockDim.x（当前块里面的线程数量）+threadIdx.x（当前线程在块中的ID）。
//        int idx = threadIdx.x + blockDim.x * blockIdx.x;
//        if (idx >= num_elem) return;
//        output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
//    }

//    void MishPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
//        int block_size = thread_count_;
//        //为了计算得到下一个最小能满足要求的整数结果 ， N需要加上 block_size(thread-1），再除以 thread ， 基本上属于向上取证的操作
//        int grid_size = (input_size_ * batchSize + block_size - 1) / block_size;
//        mish_kernel<<<grid_size, block_size>>>(inputs[0], output, input_size_ * batchSize);
//    }
//    int MishPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
//    {
//        //assert(batchSize == 1);
//        //GPU
//        //CUDA_CHECK(hipStreamSynchronize(stream));
//        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
//        return 0;
//    }

//    PluginFieldCollection MishPluginCreator::mFC{};
//    MishPluginCreator::MishPluginCreator()
//    {
////        mPluginAttributes.clear();

//        mFC.nbFields = 0;
//        mFC.fields = nullptr;
//    }

//    const char* MishPluginCreator::getPluginName() const
//    {
//        return "Mish_TRT";
//    }
//    const char* MishPluginCreator::getPluginVersion() const
//    {
//        return "1";
//    }
//    const PluginFieldCollection* MishPluginCreator::getFieldNames()
//    {
//        return &mFC;
//    }

//    IPluginV2IOExt* MishPluginCreator::createPlugin(const char* name , const PluginFieldCollection* fc)
//    {
//        MishPlugin* obj  =  new MishPlugin();
//        obj->setPluginNamespace(name);
//        return obj;
//    }
//    IPluginV2IOExt* MishPluginCreator::deserializePlugin(const char* name ,const void* seriaData , size_t seriaLen)
//    {
//        MishPlugin* obj  =  new MishPlugin(seriaData , seriaLen);
//        obj->setPluginNamespace(mNamespace.c_str());
//        return obj;
//    }
//}
#include <cmath>
#include <stdio.h>
#include <cassert>
#include <iostream>
#include "mish.h"

namespace nvinfer1
{
    MishPlugin::MishPlugin()
    {
    }

    MishPlugin::~MishPlugin()
    {
    }

    // create the plugin at runtime from a byte stream
    MishPlugin::MishPlugin(const void* data, size_t length)
    {
        assert(length == sizeof(input_size_));
        input_size_ = *reinterpret_cast<const int*>(data);
    }

    void MishPlugin::serialize(void* buffer) const
    {
        *reinterpret_cast<int*>(buffer) = input_size_;
    }

    size_t MishPlugin::getSerializationSize() const
    {
        return sizeof(input_size_);
    }

    int MishPlugin::initialize()
    {
        return 0;
    }

    Dims MishPlugin::getOutputDimensions(int index, const Dims* inputs, int nbInputDims)
    {
        assert(nbInputDims == 1);
        assert(index == 0);
        input_size_ = inputs[0].d[0] * inputs[0].d[1] * inputs[0].d[2];
        // Output dimensions
        return Dims3(inputs[0].d[0], inputs[0].d[1], inputs[0].d[2]);
    }

    // Set plugin namespace
    void MishPlugin::setPluginNamespace(const char* pluginNamespace)
    {
        mPluginNamespace = pluginNamespace;
    }

    const char* MishPlugin::getPluginNamespace() const
    {
        return mPluginNamespace;
    }

    // Return the DataType of the plugin output at the requested index
    DataType MishPlugin::getOutputDataType(int index, const nvinfer1::DataType* inputTypes, int nbInputs) const
    {
        return DataType::kFLOAT;
    }

    // Return true if output tensor is broadcast across a batch.
    bool MishPlugin::isOutputBroadcastAcrossBatch(int outputIndex, const bool* inputIsBroadcasted, int nbInputs) const
    {
        return false;
    }

    // Return true if plugin can use input that is broadcast across batch without replication.
    bool MishPlugin::canBroadcastInputAcrossBatch(int inputIndex) const
    {
        return false;
    }

    void MishPlugin::configurePlugin(const PluginTensorDesc* in, int nbInput, const PluginTensorDesc* out, int nbOutput)
    {
    }

    // Attach the plugin object to an execution context and grant the plugin the access to some context resource.
    void MishPlugin::attachToContext(cudnnContext* cudnnContext, cublasContext* cublasContext, IGpuAllocator* gpuAllocator)
    {
    }

    // Detach the plugin object from its execution context.
    void MishPlugin::detachFromContext() {}

    const char* MishPlugin::getPluginType() const
    {
        return "Mish_TRT";
    }

    const char* MishPlugin::getPluginVersion() const
    {
        return "1";
    }

    void MishPlugin::destroy()
    {
        delete this;
    }

    // Clone the plugin
    IPluginV2IOExt* MishPlugin::clone() const
    {
        MishPlugin *p = new MishPlugin();
        p->input_size_ = input_size_;
        p->setPluginNamespace(mPluginNamespace);
        return p;
    }

    __device__ float tanh_activate_kernel(float x){return (2/(1 + expf(-2*x)) - 1);}

    __device__ float softplus_kernel(float x, float threshold = 20) {
        if (x > threshold) return x;                // too large
        else if (x < -threshold) return expf(x);    // too small
        return logf(expf(x) + 1);
    }

    __global__ void mish_kernel(const float *input, float *output, int num_elem) {

        int idx = threadIdx.x + blockDim.x * blockIdx.x;
        if (idx >= num_elem) return;

        //float t = exp(input[idx]);
        //if (input[idx] > 20.0) {
        //    t *= t;
        //    output[idx] = (t - 1.0) / (t + 1.0);
        //} else {
        //    float tt = t * t;
        //    output[idx] = (tt + 2.0 * t) / (tt + 2.0 * t + 2.0);
        //}
        //output[idx] *= input[idx];
        output[idx] = input[idx] * tanh_activate_kernel(softplus_kernel(input[idx]));
    }

    void MishPlugin::forwardGpu(const float *const * inputs, float* output, hipStream_t stream, int batchSize) {
        int block_size = thread_count_;
        int grid_size = (input_size_ * batchSize + block_size - 1) / block_size;
        mish_kernel<<<grid_size, block_size>>>(inputs[0], output, input_size_ * batchSize);
    }

    int MishPlugin::enqueue(int batchSize, const void*const * inputs, void** outputs, void* workspace, hipStream_t stream)
    {
        //assert(batchSize == 1);
        //GPU
        //CUDA_CHECK(hipStreamSynchronize(stream));
        forwardGpu((const float *const *)inputs, (float*)outputs[0], stream, batchSize);
        return 0;
    }

    PluginFieldCollection MishPluginCreator::mFC{};
    std::vector<PluginField> MishPluginCreator::mPluginAttributes;

    MishPluginCreator::MishPluginCreator()
    {
        mPluginAttributes.clear();

        mFC.nbFields = mPluginAttributes.size();
        mFC.fields = mPluginAttributes.data();
    }

    const char* MishPluginCreator::getPluginName() const
    {
            return "Mish_TRT";
    }

    const char* MishPluginCreator::getPluginVersion() const
    {
            return "1";
    }

    const PluginFieldCollection* MishPluginCreator::getFieldNames()
    {
            return &mFC;
    }

    IPluginV2IOExt* MishPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc)
    {
        MishPlugin* obj = new MishPlugin();
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

    IPluginV2IOExt* MishPluginCreator::deserializePlugin(const char* name, const void* serialData, size_t serialLength)
    {
        // This object will be deleted when the network is destroyed, which will
        // call MishPlugin::destroy()
        MishPlugin* obj = new MishPlugin(serialData, serialLength);
        obj->setPluginNamespace(mNamespace.c_str());
        return obj;
    }

}

